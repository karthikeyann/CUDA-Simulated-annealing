#include "hip/hip_runtime.h"
#define N 10
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<string.h>
#include<math.h>
#include"functionpacking.cu"
#define ERROR(x) {printf(x); exit(0);}
int main(int argc, char* argv[])
{
	//Read blocks file
	char filename[30];
	sprintf(filename,"SOFT/n%d.cblks",N);
	FILE *f;
	int blkcount=0,hb,sb;
	f=fopen(filename,"r");
	if(f==NULL) ERROR("Can't open Block file\n");
	fscanf(f,"%d %d\n",&sb,&hb);
	printf("%d,%d\n",sb,hb);
	if(filename[0]=='S')	blkcount=sb;
	else blkcount=hb;
	float *w,*h,*area;
	float totalarea=0,aspectratio1,aspectratio2,aspectratio;
	w=(float*)malloc(sizeof(float)*blkcount);
	h=(float*)malloc(sizeof(float)*blkcount);
	area=(float*)malloc(sizeof(float)*blkcount);
	if(filename[0]=='S')
	{
		for(int i=0;i<blkcount;i++)
			{
			fscanf(f,"%f %f %f\n",&area[i],&aspectratio1,&aspectratio2);
			//if(rand()<RAND_MAX/2)
			// aspectratio = (aspectratio1+aspectratio2)/2.0;	//!Average
			//else
			aspectratio = aspectratio1;	//!Average
			w[i]=sqrt(area[i]*aspectratio);
			h[i]=sqrt(area[i]/aspectratio);
			totalarea=totalarea+area[i];
			}
	}
	else
	{
		for(int i=0;i<blkcount;i++)
			{
			fscanf(f,"%f %f %f\n",&area[i],&w[i],&h[i]);
			totalarea=totalarea+area[i];
			}
	}
	fclose(f);
	//Read Permutations
	int n=blkcount,s=blkcount;
	int *permutations;//[s][n];
	permutations=(int*)malloc(sizeof(int)*n*s);
	sprintf(filename,"n%ds%d.perms",N,N);
	f=fopen(filename,"r");
	if(f==NULL) ERROR("Can't open Perm file\n");
	for(int i=0;i<s;i++)
		for(int j=0;j<n;j++)
			fscanf(f,"%d",&permutations[i*n+j]);
	fclose(f);
	for(int i=0;i<s;i++){
		for(int j=0;j<n;j++)
			printf("%d ",permutations[i*n+j]);
			printf("\n");
	}
	//*/HOST Lencalc test
	float xpos[N];
	printf("Host Wid=%f\n",lencalc(permutations, permutations, w, xpos, 1));
	//*/
	//*/
	float *returnArea;
	float *d_w, *d_h, *d_returnArea;
	int *d_perms, *h_perms;
	long *seed;
	returnArea=(float*)malloc(sizeof(float)*N*N);
	h_perms=(int*)malloc(sizeof(int)*n*s);
	/*/DATA to COPY to GPU
	w n*float
	h n*float
	permutations n*s*int
	totalarea
	//*/
	hipMalloc((void**)&d_w,sizeof(float)*N);
	hipMalloc((void**)&d_h,sizeof(float)*N);
	hipMalloc((void**)&d_returnArea,sizeof(float)*N*N);
	hipMalloc((void**)&d_perms, sizeof(int)*n*s);
	hipMalloc((void**)&seed,sizeof(long)*N*N);
	
	hipMemset(seed,rand(),sizeof(long)*N*N);
	hipMemcpy(d_w,w,sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_h,h,sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_perms,permutations,sizeof(int)*n*s, hipMemcpyHostToDevice);
	annealing<<<N,N>>>(d_w, d_h, d_perms , seed, totalarea, d_returnArea);
	hipMemcpy(returnArea,d_returnArea,sizeof(float)*N*N, hipMemcpyDeviceToHost);
	printf("\n Area GPU calc\n");
	float mini=99999999; 	for(int i=0;i<N*N;i++) mini=mini<returnArea[i]?mini:returnArea[i];
	for(int i=0;i<N*N;i++)
		printf("%f, ",returnArea[i]);
	printf("\n");
	printf("Minim=%f\n",mini);
	printf("TotlA=%f\n",totalarea);
	printf("Deads=%f\n",(mini-totalarea)*100.0/mini); //Deads=8.268910 for n10s10, Deads=8.085245, time=2.440s
	hipMemcpy(h_perms,d_perms,sizeof(int)*n*s, hipMemcpyDeviceToHost);
		printf("\n Test\n");
	for(int i=0;i<s;i++){
		for(int j=0;j<n;j++)
			printf("%d ",h_perms[i*n+j]);
		printf("\n");
	}
	hipFree(d_w);
	hipFree(d_h);
	hipFree(d_returnArea);
	hipFree(d_perms);
	hipFree(seed);
	free(w); free(h); free(area); free(permutations); free(returnArea); free(h_perms);
	printf("%d %d %d\n\n\t",blkcount,n,s);
	return 0;
}
