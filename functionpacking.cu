#include "hip/hip_runtime.h"
#define FALSE 0
#define TRUE  1
__global__ void rand_init(long *a,long seed)
{
int tid=threadIdx.x+blockDim.x*blockIdx.x;
//long a = 100001;
a[tid] = seed + tid;
}
// returns random integer from 1 to lim
__device__ int rand1(long *a, int lim)
{
int tid=threadIdx.x+blockDim.x*blockIdx.x;
a[tid] = (a[tid] * 125) % 2796203;
return ((a[tid] % lim) + 1);
}
// returns random integer from 1 to lim (Gerhard's generator)
__device__ int rand2(long *a, int lim)
{
int tid=threadIdx.x+blockDim.x*blockIdx.x;
a[tid] = (a[tid] * 32719 + 3) % 32749;
return ((a[tid] % lim) + 1);
}
// returns random integer from 1 to lim (Bill's generator)
__device__ int rand3(long *a, int lim)
{
int tid=threadIdx.x+blockDim.x*blockIdx.x;
a[tid] = (((a[tid] * 214013L + 2531011L) >> 16) & 32767);
return ((a[tid] % lim) + 1);
}

__host__ __device__ float lencalc(int *sx, int *sy, float *w, float *xpos, bool axis)
{
	//int cost=0;
	auto int matchy[N];//,matchx[N];
	auto float L[N];
	for(int i=0;i<N;i++) {
	 //matchx[sx[i]-1]=i;
	 matchy[sy[i]-1]=i;
	 L[i]=0; xpos[i]=0;
	  }
	int b,p;
	float t;
	for(int i=0;i<N;i++){
		if(axis) b=sx[i]-1;
		else b=sx[N-1-i]-1;
		p=matchy[b];
		xpos[b]=L[p];
		t=xpos[b]+w[b];
		for(int j=p; j<N && t>L[j] ;j++) L[j]=t;
	}
	return L[N-1];
}
__device__ void neighbourhood(int *sx,int *sy, int *sxnew, int *synew, long *seed){
	int randxi=rand1(seed,N-1),randxj=rand1(seed,N-1); //In 1D gridding, use N-1,N. for 2D gridding use N-1,N-1	
	int randyi=rand1(seed,N  ),randyj=rand1(seed,N  ); //for 1D N, 2D N-1
	//swap
	for(int i=0;i<N;i++){
		sxnew[i]=sx[i]; synew[i]=sy[i];
	}
	sxnew[randxi]=sx[randxj];
	sxnew[randxj]=sx[randxi];
	synew[randxi]=sy[randxj];
	synew[randxj]=sy[randxi];
}
__device__ void newneighbourhood(int *sx,int *sy, int *sxnew, int *synew, long *seed){
	int randxi=rand1(seed,N-1),randxj=rand1(seed,N-1); //In 1D gridding, use N-1,N. for 2D gridding use N-1,N-1	
	int randyi=rand1(seed,N  ),randyj=rand1(seed,N  ); //for 1D N, 2D N-1
	int choice=rand1(seed,3  );
	for(int i=0;i<N;i++){
		sxnew[i]=sx[i]; synew[i]=sy[i];
	}
	if(choice==1){
		//swap sx
		sxnew[randxi]=sx[randxj];
		sxnew[randxj]=sx[randxi];		
	}
	else if(choice==2){
		synew[randxi]=sy[randxj];
		synew[randxj]=sy[randxi];
	}
	else{
		sxnew[randxi]=sx[randxj];
		sxnew[randxj]=sx[randxi];
		synew[randxi]=sy[randxj];
		synew[randxj]=sy[randxi];
	}
}

__global__ void annealing(float *w, float *h, int *perms , long *seed, int totalarea, float *returnArea){
	int tid=threadIdx.x, uid=blockIdx.x;
	int index1d=threadIdx.x+blockDim.x*blockIdx.x;
	__shared__ float minarea[N];
	__shared__ int minid[N];
	auto int sxA[N], syA[N], sxnewA[N], synewA[N]; //Arrays
	auto int *sx=sxA, *sy=syA, *sxnew=sxnewA, *synew=synewA; //pointer seperately declared to tackle lvalue error
	for(int i=0;i<N;i++){
		sx[i]=perms[uid*N+i];
		sy[i]=perms[tid*N+i]; //use uid, if 1D grid, else 2D grid
		//perms[uid*N+i]=-sy[i];
	}
	float t0=0.00000001, ts=0.01,T,delta; //dead space %4.868905504 in python, %8.26 in CUDA.
	int accepted=0;
	float Wid=0,Len=0,Area=0;
	auto float xpos[N],ypos[N];
	float sdWid=0,sdLen=0,sdArea=0;
	float minsofar=0;
	Wid=lencalc(sx,sy,w,xpos,FALSE); //X-axis
	Len=lencalc(sx,sy,h,ypos,TRUE ); //Y-axis
	Area=Wid*Len;
	returnArea[index1d]=Area;
	for(int ancnt=0;ancnt<5000;ancnt++) //20000
	{
	newneighbourhood(sx,sy,sxnew,synew,seed);
	sdWid=lencalc(sxnew,synew,w,xpos,FALSE); //X-axis
	sdLen=lencalc(sxnew,synew,h,ypos,TRUE ); //Y-axis
	sdArea=sdWid*sdLen;

	auto int *temp;
	if(sdArea<Area){
		temp=sx;
		sx=sxnew;
		sxnew=temp;
		temp=sy;
		sy=synew;
		synew=temp;
		Area=sdArea;
		accepted=accepted+1;
		minsofar=Area;
	}
	else{ //take risk (uses random fn)
		float p= float(rand1(seed,N))/float(N);
		T = 1.0/(t0+ts*accepted);
		delta=sdArea-Area;
		if(p<exp(-delta/T)){
			temp=sx;
			sx=sxnew;
			sxnew=temp;
			temp=sy;
			sy=synew;
			synew=temp;
			Area=sdArea;
			accepted=accepted+1;
		}
	}
	}
	minarea[threadIdx.x]=minsofar;
	minid[threadIdx.x]=threadIdx.x;
	for(int ruled=N/2;ruled>0;ruled/=2)
		if(threadIdx.x<ruled)
			if(minarea[threadIdx.x]>minarea[threadIdx.x+ruled]){
				minarea[threadIdx.x]=minarea[threadIdx.x+ruled];
				minid[threadIdx.x]=minid[threadIdx.x+ruled];
				}
	if(threadIdx.x==minid[0])
	for(int i=0;i<N;i++){
		perms[uid*N+i]=sx[i];
		//perms[uid*N+i]=sy[i]; //use uid, if 1D grid, else tid for 2D grid 
	}
	returnArea[index1d]=minsofar;
}
